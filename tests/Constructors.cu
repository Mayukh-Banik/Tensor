#include "../Tensor/core/Tensor.cuh"

#include <gtest/gtest.h>

#define DATA_TYPE double

template <typename T>
int arePropsValid(Tensor<T> &t, uint64_t len, uint64_t ndim, uint64_t *shape,
				  uint64_t *strides, uint64_t elementCount, bool Alloc = true)
{
	if (Alloc == true)
	{
		if (t.buf == NULL)
		{
			return 1;
		}
	}
	if (t.len != len)
	{
		return 2;
	}
	if (t.ndim != ndim)
	{
		return 3;
	}
	for (uint64_t i = 0; i < ndim; i++)
	{
		if (t.strides[i] != strides[i] || t.shape[i] != shape[i])
		{
			return 4;
		}
	}
	if (t.elementCount != elementCount)
	{
		return 5;
	}
	return 0;
}

TEST(Defualt_Constructor, heapAllocation)
{
	Tensor<DATA_TYPE> *x;
	EXPECT_NO_THROW(x = new Tensor<DATA_TYPE>());
	EXPECT_NO_THROW(delete (x));
}

TEST(Defualt_Constructor, stackAllocation)
{
	EXPECT_NO_THROW(Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>());
}

TEST(Defualt_Constructor, getZero)
{
	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>();
	EXPECT_ANY_THROW(x.getIndex(0));
}

TEST(Defualt_Constructor, setZero)
{
	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>();
	EXPECT_ANY_THROW(x.setIndex(0, 0));
}

TEST(Defualt_Constructor, properties)
{
	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>();
	EXPECT_EQ(0, arePropsValid(x, 0, 0, NULL, NULL, 0, false));
}

TEST(Scalar_Constructor, heapAllocation)
{
	Tensor<DATA_TYPE> *x;
	EXPECT_NO_THROW(x = new Tensor<DATA_TYPE>(1));
	EXPECT_NO_THROW(delete (x));
}

TEST(Scalar_Constructor, stackAllocation)
{
	EXPECT_NO_THROW(Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(2));
}

TEST(Scalar_Constructor, inBoundSetting)
{
	DATA_TYPE y = 2;
	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(y);
	EXPECT_NO_THROW(x.getIndex(0));
	EXPECT_EQ(y, x.getIndex(0));
	EXPECT_ANY_THROW(x.getIndex(1));
	EXPECT_NO_THROW(x.setIndex(0, 1));
	EXPECT_EQ(3, x.setIndex(0, 3));
	EXPECT_ANY_THROW(x.setIndex(5, 5));
}

TEST(Scalar_Constructor, properties)
{
	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(1);
	EXPECT_EQ(0, arePropsValid(x, sizeof(DATA_TYPE), 0, NULL, NULL, 1));
}

TEST(ShapeNdim_Constructor, heapAllocation)
{
	uint64_t shape[] = {3, 4};
	Tensor<DATA_TYPE> *x;
	EXPECT_NO_THROW(x = new Tensor<DATA_TYPE>(shape, 2));
	EXPECT_NO_THROW(delete (x));
}

TEST(ShapeNdim_Constructor, stackAllocation)
{
	uint64_t shape[] = {2, 5};
	EXPECT_NO_THROW(Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(shape, 2));
}

TEST(ShapeNdim_Constructor, properties)
{
	uint64_t shape[] = {3, 4};
	uint64_t strides[] = {4 * sizeof(DATA_TYPE), sizeof(DATA_TYPE)};
	uint64_t ndim = 2;
	uint64_t elementCount = 3 * 4;
	uint64_t len = elementCount * sizeof(DATA_TYPE);

	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(shape, ndim);
	EXPECT_EQ(0, arePropsValid(x, len, ndim, shape, strides, elementCount));
}

TEST(ShapeNdim_Constructor, invalidAccess)
{
	uint64_t shape[] = {3, 3};
	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(shape, 2);
	EXPECT_ANY_THROW(x.getIndex(9));
	EXPECT_NO_THROW(x.setIndex(0, 1.5));
	EXPECT_ANY_THROW(x.setIndex(10, 2.5));
}

TEST(Vector_Shape_Constructor, heapAllocation)
{
	std::vector<uint64_t> shape = {3, 4};
	Tensor<DATA_TYPE> *x;
	EXPECT_NO_THROW(x = new Tensor<DATA_TYPE>(shape));
	EXPECT_NO_THROW(delete (x));
}

TEST(Vector_Shape_Constructor, stackAllocation)
{
	std::vector<uint64_t> shape = {2, 5};
	EXPECT_NO_THROW(Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(shape));
}

TEST(Vector_Shape_Constructor, properties)
{
	std::vector<uint64_t> shape = {3, 4};
	uint64_t strides[] = {4 * sizeof(DATA_TYPE), sizeof(DATA_TYPE)};
	uint64_t ndim = shape.size();
	uint64_t elementCount = 3 * 4;
	uint64_t len = elementCount * sizeof(DATA_TYPE);

	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(shape);
	EXPECT_EQ(0, arePropsValid(x, len, ndim, shape.data(), strides, elementCount));
}

TEST(Vector_Shape_Constructor, invalidAccess)
{
	std::vector<uint64_t> shape = {3, 3};
	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(shape);
	EXPECT_ANY_THROW(x.getIndex(9));
	EXPECT_NO_THROW(x.setIndex(0, 1.5));
	EXPECT_ANY_THROW(x.setIndex(10, 2.5));
}

TEST(Raw_Pointer_Shape_Array_Constructor, heapAllocation)
{
	DATA_TYPE values[] = {1, 2, 3, 4};
	uint64_t shape[] = {2, 2};
	Tensor<DATA_TYPE> *x;
	EXPECT_NO_THROW(x = new Tensor<DATA_TYPE>(values, shape, 2));
	EXPECT_NO_THROW(delete x);
}

TEST(Raw_Pointer_Shape_Array_Constructor, stackAllocation)
{
	DATA_TYPE values[] = {1, 2, 3, 4};
	uint64_t shape[] = {2, 2};
	EXPECT_NO_THROW(Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(values, shape, 2));
}

TEST(Raw_Pointer_Shape_Array_Constructor, properties)
{
	DATA_TYPE values[] = {1, 2, 3, 4};
	uint64_t shape[] = {2, 2};
	uint64_t strides[] = {2 * sizeof(DATA_TYPE), sizeof(DATA_TYPE)};
	uint64_t ndim = 2;
	uint64_t elementCount = 4;
	uint64_t len = elementCount * sizeof(DATA_TYPE);

	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(values, shape, ndim);
	EXPECT_EQ(0, arePropsValid(x, len, ndim, shape, strides, elementCount));

	for (uint64_t i = 0; i < elementCount; i++)
	{
		EXPECT_EQ(values[i], x.getIndex(i));
	}
}

TEST(Raw_Pointer_Vector_Shape_Constructor, heapAllocation)
{
	DATA_TYPE values[] = {1, 2, 3, 4};
	std::vector<uint64_t> shape = {2, 2};
	Tensor<DATA_TYPE> *x;
	EXPECT_NO_THROW(x = new Tensor<DATA_TYPE>(values, shape));
	EXPECT_NO_THROW(delete x);
}

TEST(Raw_Pointer_Vector_Shape_Constructor, stackAllocation)
{
	DATA_TYPE values[] = {1, 2, 3, 4};
	std::vector<uint64_t> shape = {2, 2};
	EXPECT_NO_THROW(Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(values, shape));
}

TEST(Cuda_Raw_Pointer_Shape_Array_Constructor, heapAllocation)
{
	const DATA_TYPE values[] = {1.0, 2.0, 3.0, 4.0};
	const uint64_t shape[] = {2, 2};
	Tensor<DATA_TYPE> *x;
	EXPECT_NO_THROW(x = new Tensor<DATA_TYPE>(values, shape, 2));
	EXPECT_NO_THROW(delete x);
}

TEST(Cuda_Raw_Pointer_Shape_Array_Constructor, stackAllocation)
{
	const DATA_TYPE values[] = {1.0, 2.0, 3.0, 4.0};
	const uint64_t shape[] = {2, 2};
	EXPECT_NO_THROW(Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(values, shape, 2));
}

TEST(Cuda_Raw_Pointer_Shape_Array_Constructor, properties)
{
	const DATA_TYPE values[] = {1.0, 2.0, 3.0, 4.0};
	uint64_t shape[] = {2, 2};
	uint64_t strides[] = {2 * sizeof(DATA_TYPE), sizeof(DATA_TYPE)};
	uint64_t ndim = 2;
	uint64_t elementCount = 4;
	uint64_t len = elementCount * sizeof(DATA_TYPE);

	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(values, shape, ndim);
	EXPECT_EQ(0, arePropsValid(x, len, ndim, shape, strides, elementCount));

	DATA_TYPE hostBuffer[4];
	hipError_t err = hipMemcpy(hostBuffer, x.buf, len, hipMemcpyDeviceToHost);
	EXPECT_EQ(hipSuccess, err);
	for (uint64_t i = 0; i < elementCount; i++)
	{
		EXPECT_EQ(values[i], hostBuffer[i]);
	}
}

TEST(Cuda_Raw_Pointer_Vector_Shape_Constructor, heapAllocation)
{
	const DATA_TYPE values[] = {1.0, 2.0, 3.0, 4.0};
	const std::vector<uint64_t> shape = {2, 2};
	Tensor<DATA_TYPE> *x;
	EXPECT_NO_THROW(x = new Tensor<DATA_TYPE>(values, shape));
	EXPECT_NO_THROW(delete x);
}

TEST(Cuda_Raw_Pointer_Vector_Shape_Constructor, stackAllocation)
{
	const DATA_TYPE values[] = {1.0, 2.0, 3.0, 4.0};
	const std::vector<uint64_t> shape = {2, 2};
	EXPECT_NO_THROW(Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(values, shape));
}

TEST(Cuda_Raw_Pointer_Vector_Shape_Constructor, properties)
{
	const DATA_TYPE values[] = {1.0, 2.0, 3.0, 4.0};
	std::vector<uint64_t> shape = {2, 2};
	uint64_t strides[] = {2 * sizeof(DATA_TYPE), sizeof(DATA_TYPE)};
	uint64_t ndim = shape.size();
	uint64_t elementCount = 4;
	uint64_t len = elementCount * sizeof(DATA_TYPE);

	Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(values, shape);
	EXPECT_EQ(0, arePropsValid(x, len, ndim, shape.data(), strides, elementCount));

	DATA_TYPE hostBuffer[4];
	hipError_t err = hipMemcpy(hostBuffer, x.buf, len, hipMemcpyDeviceToHost);
	EXPECT_EQ(hipSuccess, err);
	for (uint64_t i = 0; i < elementCount; i++)
	{
		EXPECT_EQ(values[i], hostBuffer[i]);
	}
}

TEST(Cuda_Raw_Pointer_Vector_Shape_Constructor, emptyShape)
{
	const DATA_TYPE values[] = {1.0};
	const std::vector<uint64_t> emptyShape = {};
	EXPECT_ANY_THROW(Tensor<DATA_TYPE> x = Tensor<DATA_TYPE>(values, emptyShape));
}

